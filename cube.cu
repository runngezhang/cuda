#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void cube(float *d_out, float* d_in)
{
	int idx = threadIdx.x;
	float f = d_in[idx] ;
	//The d_ implies that the array sits on the device
	d_out[idx] = f*f*f;
}//end of kernel 'cube'

int main(int argc, char** argv)
{
	const int ARRAY_SIZE = 96;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	//generate input array on the host
	float h_in[ARRAY_SIZE];
	//the h_ implies that the array sits on the host
	//Build the test array - the elements of which are going to be cubed
	for(int i=0; i< ARRAY_SIZE; i++)
	{
		h_in[i] = float(i);
	}//end of for
	float h_out[ARRAY_SIZE];
	
	//declare GPU memory pointers
	float *d_in;
	float *d_out;

	//allocate memory for the two arrays on the device
	hipMalloc((void**)&d_in,ARRAY_BYTES);
	hipMalloc((void**)&d_out,ARRAY_BYTES);
	
	//transfer the array to the GPU
	// destination,source,size,method
	hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);

	//launch the kernel	
	cube<<<1,ARRAY_SIZE>>>(d_out,d_in);
	//Kernelname<<<number_of_blocks,number_of_threads_per_block>>(parameters to kernel);

	//copy the results back onto the device
	hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);
	
	//print out the results
	for(int i=0;i < ARRAY_SIZE; i++)
	{
		cout<<i<<":"<<h_out[i]<<endl;
	}//end of for
	
	hipFree(d_in);
	hipFree(d_out);

}//end of main
